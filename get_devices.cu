#include <hip/hip_runtime.h>
#include <stdio.h>

int main(){
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) printf("No cuda device found\n");
    else {
        printf("Found %d devices\n", deviceCount);
        hipDeviceProp_t pr;
        for (int i=0; i<deviceCount; i++){
            hipGetDeviceProperties(&pr, i);
            printf("Device Number: %d\n", i);
            printf("  Device name: %s\n", pr.name);
            printf("  Memory Clock Rate (MHz): %d\n", pr.memoryClockRate/1024);
            printf("  Memory Bus Width (bits): %d\n", pr.memoryBusWidth);
            printf("  Peak Memory Bandwidth (GB/s): %.1f\n", 2.0*pr.memoryClockRate*(pr.memoryBusWidth/8)/1.0e6);
            printf("  Total global memory (Gbytes) %.1f\n",(float)(pr.totalGlobalMem)/1024.0/1024.0/1024.0);
            printf("  Shared memory per block (Kbytes) %.1f\n",(float)(pr.sharedMemPerBlock)/1024.0);
            printf("  minor-major: %d-%d\n", pr.minor, pr.major);
            printf("  Warp-size: %d\n", pr.warpSize);
            printf("  Concurrent kernels: %s\n", pr.concurrentKernels ? "yes" : "no");
            printf("  Concurrent computation/communication: %s\n\n",pr.deviceOverlap ? "yes" : "no");

            printf("  Max Thread per block: %d\n", pr.maxThreadsPerBlock);
            printf("  Max Thread per multiprocessor: %d\n", pr.maxThreadsPerMultiProcessor);
            printf("  Max Thread Multi processor Count: %d\n", pr.multiProcessorCount);
            printf("  Multi processor count: %d\n\n", pr.multiProcessorCount);
        }
    }
}
